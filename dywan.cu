// Compile: nvcc dywan.cu -o cu_dywan

#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cmath>

struct box
{
    int x0; // pixel startowy w osi x
    int xn; // pixel koncowy w osi x
    int y0; // pixel startowy w osi y
    int yn; // pixel koncowy w osi y

    int id; // id pixela, między 0 a 8 - służy do stwierdzenia czy pixel jest czarny czy biały // czy dalej wchodzić w rekursje
};

#define WIDTH (9 * 9 * 9 * 9 * 9)
#define HEIGHT (9 * 9 * 9 * 9 * 9)

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void sierpinskiKernel(box *boxesIn, box *boxesOut, int *pixelArray, int numBoxes, int currentDepth)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numBoxes)
        return;

    box b = boxesIn[idx];

    int width = b.xn - b.x0;
    int height = b.yn - b.y0;

    if (b.id == 4)
    {
        for (int x = b.x0; x < b.xn; x++)
        {
            for (int y = b.y0; y < b.yn; y++)
            {
                pixelArray[y * WIDTH + x] = 255;
            }
        }
        return;
    }
    else if (width < 2 || height < 2)
    {
        for (int x = b.x0; x < b.xn; x++)
        {
            for (int y = b.y0; y < b.yn; y++)
            {
                pixelArray[y * WIDTH + x] = 0;
            }
        }
        return;
    }

    // podział na 9 nowych boksów
    int dx = width / 3;
    int dy = height / 3;
    for (int i = 0; i < 9; i++)
    {
        int col = i % 3;
        int row = i / 3;

        int outIdx = idx * 9 + i; // bo każdy wątek tworzy 9 boksów

        boxesOut[outIdx].x0 = b.x0 + col * dx;
        boxesOut[outIdx].xn = b.x0 + (col + 1) * dx;
        boxesOut[outIdx].y0 = b.y0 + row * dy;
        boxesOut[outIdx].yn = b.y0 + (row + 1) * dy;
        boxesOut[outIdx].id = i;
    }
}

int main(void)
{
    const int maxDepth = 5; // zależnie od WIDTH/HEIGHT
    box *d_current;
    box *d_next;

    box startBox;
    startBox.x0 = 0;
    startBox.xn = WIDTH;
    startBox.y0 = 0;
    startBox.yn = HEIGHT;
    startBox.id = -1;

    hipMalloc(&d_current, sizeof(box) * 1);
    hipMemcpy(d_current, &startBox, sizeof(box), hipMemcpyHostToDevice);
    hipMalloc(&d_next, sizeof(box) * pow(9, maxDepth));

    int *d_pixels;
    hipMalloc(&d_pixels, sizeof(int) * WIDTH * HEIGHT);
    hipMemset(d_pixels, 0, sizeof(int) * WIDTH * HEIGHT);

    int numBoxes = 1;

    for (int depth = 0; depth < maxDepth; ++depth)
    {
        int threads = 256;
        int blocks = (numBoxes + threads - 1) / threads;

        sierpinskiKernel<<<blocks, threads>>>(d_current, d_next, d_pixels, numBoxes, depth);
        hipDeviceSynchronize();
        HANDLE_ERROR(hipGetLastError());

        HANDLE_ERROR(hipDeviceSynchronize());

        std::swap(d_current, d_next);
        numBoxes *= 9;
    }

    int *pixels = new int[WIDTH * HEIGHT];
    hipMemcpy(pixels, d_pixels, sizeof(int) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);

    hipFree(d_pixels);
    hipFree(d_current);
    hipFree(d_next);

    for (int y = 0; y < HEIGHT; y += HEIGHT / 100)
    {
        for (int x = 0; x < WIDTH; x += WIDTH / 100)
        {
            std::cout << (pixels[y * WIDTH + x] > 0 ? " " : "#");
        }
        std::cout << std::endl;
    }

    return EXIT_SUCCESS;
}